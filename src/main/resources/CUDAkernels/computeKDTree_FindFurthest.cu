
#include <hip/hip_runtime.h>
extern "C"

/*
                    Pointer.to(gDots.iGA_Float[GPUDots.PX].gpuArray), 
                    Pointer.to(gDots.iGA_Float[GPUDots.PY].gpuArray),
                    Pointer.to(gDots.iGA_Float[GPUDots.PZ].gpuArray),
                    // Blocks Properties
                    Pointer.to(iGA_arrayDotsIndexes.gpuArray),
                    Pointer.to(iGA_addrStartBlock0.gpuArray),Pointer.to(iGA_nPtBlock0.gpuArray),
                    Pointer.to(iGA_addrStartBlock1.gpuArray),Pointer.to(iGA_nPtBlock1.gpuArray),
                    Pointer.to(avgX.gpuArray), Pointer.to(avgY.gpuArray),Pointer.to(avgZ.gpuArray),
                    Pointer.to(iGA_idBlock.gpuArray),
                    Pointer.to(iGA_offsIntBlock.gpuArray),   
                    // Output values
                    Pointer.to(idPtFar.gpuArray),
                    Pointer.to(distPtFar.gpuArray)
*/

__global__ void findFurthest(// Dots props
						   float* pX,
                           float* pY,
                           float* pZ,
						   //Tree specs
						   // per Block
						   int* dotIndexes, 
                           int* stBl0, int* nPtBl0,
						   int* stBl1, int* nPtBl1,
						   float* avgPX, 
                           float* avgPY,
                           float* avgPZ,
						   // per GPU Block
						   int* idBl, int* offsBl,
						   
						   // output values, per block
						   int* idFurthest, float* dMax
								/*float* pX,float* pY,float* pZ, 
                                 float* avgPX, float* avgPY, float* avgPZ,
                                 int* lockBlock, float* dMax, 
                                 int* idFurthest,
                                 int* id_in, int* id_bl_in*/
								 )
{
        // Put data in shared memory of the block
    //extern __shared__ float dMax_intermed[];
    //extern __shared__ int indexMax_intermed[];
    
    extern __shared__ int array[];    
    float* posAVGBlock = (float*)&array[5]; 
    float* dMaxPt = (float*)&posAVGBlock[3];
    int*   iMaxPt =   (int*)&dMaxPt[blockDim.x];    
    
	// Fetch block data
    int iGPUBlock=blockIdx.x;
	int iThread=threadIdx.x;
	int idBloc;
	if (iThread==0) {
		 idBloc=idBl[iGPUBlock];		 
		 array[0]=offsBl[iGPUBlock];
		 array[1]=stBl0[idBloc];
		 array[2]=nPtBl0[idBloc];
		 array[3]=stBl1[idBloc];		 
		 array[4]=nPtBl1[idBloc];
		 posAVGBlock[0]=avgPX[idBloc];
		 posAVGBlock[1]=avgPY[idBloc];
		 posAVGBlock[2]=avgPZ[idBloc];
	}
	__syncthreads();	
	
	int offsPt = array[0];
	int startIndexBl0 = array[1];	 
	int nPtBlock0 = array[2];
	int startIndexBl1 = array[3]; // useless in fact
	int nPtBlock1 = array[4];
	int nPts = nPtBlock0 + nPtBlock1;
	int ptToBeComputed = iThread+offsPt;
	int mx=posAVGBlock[0];
	int my=posAVGBlock[1];
	int mz=posAVGBlock[2];

	if (ptToBeComputed<nPts) {
		int id_pt=dotIndexes[startIndexBl0+ptToBeComputed];
		float xval=(pX[id_pt]-mx);
        float yval=(pY[id_pt]-my);
        float zval=(pZ[id_pt]-mz);                
        dMaxPt[iThread]=xval*xval+yval*yval+zval*zval;
        iMaxPt[iThread]=id_pt;
	} else {
		dMaxPt[iThread]=-1;
        iMaxPt[iThread]=-1;
	}       
    __syncthreads();
    // All data copied to shared Mem
         
    for (unsigned int s=blockDim.x/2;s>0;s>>=1)
    {
        if (iThread<s) {       
            int tShift=s;
            float dTest=dMaxPt[iThread+tShift];
            if (dTest>=dMaxPt[iThread]) {
                dMaxPt[iThread]=dTest;
                iMaxPt[iThread]=iMaxPt[iThread+tShift];  
            }
        }
         __syncthreads();
    }
          
          if (iThread==0) {
                 //atomicAdd(& sigX[id_bloc], posdata_add[cPos]);
                 //atomicAdd(& sigY[id_bloc], posdata_add[cPos+1]);
                 //atomicAdd(& sigZ[id_bloc], posdata_add[cPos+2]);
                 float dMaxBlock=dMaxPt[0];
                 int iMaxBlock=iMaxPt[0];
                 //printf("iMaxBlock = %i; dMaxBloc = %f; GPUBlock = %i; id_bloc = %i \n", iMaxBlock, dMaxBlock, blockIdx.x, id_bloc);
                 
                 // We got a problem with this critical section! I remove it.
                  
                 //while (atomicCAS(& lockBlock[id_bloc], 0, 1) != 0); // Wait to take the semaphore
                    // read current Max
                float dTest=dMax[idBloc];
                    if (dMaxBlock>=dTest) {
                         dMax[idBloc]=dMaxBlock;
                         idFurthest[idBloc]=iMaxBlock;
                    }                    // proxy to avoid long locks... maybe can be improved with kernel map then kernel reduce
                 //lockBlock[id_bloc]=0; // Release the semaphore
                 //__threadfence();
          }         
     
    
}
