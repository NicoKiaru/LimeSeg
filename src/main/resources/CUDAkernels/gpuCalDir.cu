
#include <hip/hip_runtime.h>
extern "C"

__global__ void calcDir(// Dots props
						   float* pX,
                           float* pY,
                           float* pZ,
						   //Tree specs
						   // per Block
						   int* dotIndexes, 
                           int* stBl0, int* nPtBl0,
						   int* stBl1, int* nPtBl1,
						   float* avgPX, 
                           float* avgPY,
                           float* avgPZ,
						   // per GPU Block
						   int* idBl, int* offsBl,
						   
						   // output values, per block
						   int* idFurthest, float* dMax
								/*float* pX,float* pY,float* pZ, 
                                 float* avgPX, float* avgPY, float* avgPZ,
                                 int* lockBlock, float* dMax, 
                                 int* idFurthest,
                                 int* id_in, int* id_bl_in*/
								 )
{   
    extern __shared__ int array[];    
    float* posAVGBlock = (float*)&array[5]; 
    float* dMaxPt = (float*)&posAVGBlock[3];
    int*   iMaxPt =   (int*)&dMaxPt[blockDim.x];    
    
	// Fetch block data
    int iGPUBlock=blockIdx.x;
	int iThread=threadIdx.x;
	int idBloc;
	if (iThread==0) {
		 idBloc=idBl[iGPUBlock];		 
		 array[0]=offsBl[iGPUBlock];
		 array[1]=stBl0[idBloc];
		 array[2]=nPtBl0[idBloc];
		 array[3]=stBl1[idBloc];		 
		 array[4]=nPtBl1[idBloc];
		 posAVGBlock[0]=avgPX[idBloc];
		 posAVGBlock[1]=avgPY[idBloc];
		 posAVGBlock[2]=avgPZ[idBloc];
	}
	__syncthreads();	
	
	int offsPt = array[0];
	int startIndexBl0 = array[1];	 
	int nPtBlock0 = array[2];
	int startIndexBl1 = array[3]; // useless in fact
	int nPtBlock1 = array[4];
	int nPts = nPtBlock0 + nPtBlock1;
	int ptToBeComputed = iThread+offsPt;
	int mx=posAVGBlock[0];
	int my=posAVGBlock[1];
	int mz=posAVGBlock[2];

	if (ptToBeComputed<nPts) {
		int id_pt=dotIndexes[startIndexBl0+ptToBeComputed];
		float xval=(pX[id_pt]-mx);
        float yval=(pY[id_pt]-my);
        float zval=(pZ[id_pt]-mz);                
        dMaxPt[iThread]=xval*xval+yval*yval+zval*zval;
        iMaxPt[iThread]=id_pt;
	} else {
		dMaxPt[iThread]=-1;
        iMaxPt[iThread]=-1;
	}       
    __syncthreads();
    // All data copied to shared Mem
         
    
}
